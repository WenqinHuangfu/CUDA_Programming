#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* To save you time, we are including all 6 variants of the loop ordering
   as separate functions and then calling them using function pointers.
   The reason for having separate functions that are nearly identical is
   to avoid counting any extraneous processing towards the computation
   time.  This includes I/O accesses (printf) and conditionals (if/switch).
   I/O accesses are slow and conditional/branching statements could
   unfairly bias results (lower cases in switches must run through more
   case statements on each iteration).
*/

const int TILE_WIDTH_GEMM = 16;
const int TILE_DIM = 32;

void multMat1( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ijk loop order. */
    for( i = 0; i < n; i++ )
        for( j = 0; j < n; j++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat2( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ikj loop order. */
    for( i = 0; i < n; i++ )
        for( k = 0; k < n; k++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat3( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jik loop order. */
    for( j = 0; j < n; j++ )
        for( i = 0; i < n; i++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat4( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jki loop order. */
    for( j = 0; j < n; j++ )
        for( k = 0; k < n; k++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat5( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kij loop order. */
    for( k = 0; k < n; k++ )
        for( i = 0; i < n; i++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat6( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kji loop order. */
    for( k = 0; k < n; k++ )
        for( j = 0; j < n; j++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

/* Question 1 */
// GPU based GEMM with SM-specific shared memory
__global__ void MatrixMultiplyKernel(const float* devM, const float* devN,float* devP, const int width){
	__shared__ float sM[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	__shared__ float sN[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = bx * TILE_WIDTH_GEMM + bx;
	int row = by * TILE_WIDTH_GEMM + ty;
	
	// Initialize accumulator to 0. Then multiply/add
	float pValue = 0;
	
	for (int m = 0; m < width / TILE_WIDTH_GEMM; m++) {
		sM[ty][tx] = devM[row *width+(m*TILE_WIDTH_GEMM + tx)];
		sN[ty][tx] = devN[col+(m *TILE_WIDTH_GEMM+ty)*width];
		__syncthreads();
		
		for (int k = 0; k < TILE_WIDTH_GEMM; ++k)
			pValue += sM[ty][k] * sN[k][tx];
		__syncthreads();
	}
	
	devP[row * width + col] = pValue;
}

/* Question 2 */
// Simple matrix copying
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// Matrix copy with shared memory
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// Native transpose
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// Coalesced transpose with block shared memory
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// Coalesced transpose with shared memory and matrix padding
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

/* uses timing features from sys/time.h that you haven't seen before */
int main( int argc, char **argv ) {
    // CPU implementation
    int nmax = 1000,i;

    void (*orderings[])(int,float *,float *,float *) =
        {&multMat1,&multMat2,&multMat3,&multMat4,&multMat5,&multMat6};
    char *names[] = {"ijk","ikj","jik","jki","kij","kji"};

    float *A = (float *)malloc( nmax*nmax * sizeof(float));
    float *B = (float *)malloc( nmax*nmax * sizeof(float));
    float *C = (float *)malloc( nmax*nmax * sizeof(float));

    struct timeval start, end;

    /* fill matrices with random numbers */
    for( i = 0; i < nmax*nmax; i++ ) A[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) B[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) C[i] = drand48()*2-1;

    for( i = 0; i < 6; i++) {
        /* multiply matrices and measure the time */
        gettimeofday( &start, NULL );
        (*orderings[i])( nmax, A, B, C );
        gettimeofday( &end, NULL );

        /* convert time to Gflop/s */
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        double Gflops = 2e-9*nmax*nmax*nmax/seconds;
        printf( "%s:\tn = %d, %.3f Gflop/s\n", names[i], nmax, Gflops );
    }
    
    free( A );
    free( B );
    free( C );
    
    printf("\n\n");
    
    // GPU implementation
    int m_size = 1600, n_size = 1600;
    int width = 1600;
    int iterations = 100;
    float GFLOPs = 0;
    
    float *A_h = (float *)malloc( m_size*n_size*sizeof(float));
    float *B_h = (float *)malloc( m_size*n_size*sizeof(float));
    float *C_h = (float *)malloc( m_size*n_size*sizeof(float));

    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, m_size*n_size*sizeof(float));
    hipMalloc((void**)&B_d, m_size*n_size*sizeof(float));
    hipMalloc((void**)&C_d, m_size*n_size*sizeof(float));
    
    dim3 dimGrid(100, 100, 1);
    dim3 dimBlock(16, 16, 1);
    
    hipMemcpy(A_d, A_h, m_size*n_size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, m_size*n_size*sizeof(float), hipMemcpyHostToDevice);
		
    for (int i = 0; i < iterations; i++) {
        gettimeofday( &start, NULL );
        MatrixMultiplyKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, width);
        gettimeofday( &end, NULL );
			
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        GFLOPs += 2e-9*width*width*width/seconds;
    }
		
    hipMemcpy(C_h, C_d, m_size*n_size*sizeof(float), hipMemcpyDeviceToHost);

    GFLOPs /= iterations;
		
    printf( "%.3f GFLOPs/s\n", GFLOPs );

    hipFree( A_d );
    hipFree( B_d );
    hipFree( C_d );
    free( A_h );
    free( B_h );
    free( C_h );

    printf("\n\n");

    return 0;
}
