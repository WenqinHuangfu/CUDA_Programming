#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <assert.h>

/* To save you time, we are including all 6 variants of the loop ordering
   as separate functions and then calling them using function pointers.
   The reason for having separate functions that are nearly identical is
   to avoid counting any extraneous processing towards the computation
   time.  This includes I/O accesses (printf) and conditionals (if/switch).
   I/O accesses are slow and conditional/branching statements could
   unfairly bias results (lower cases in switches must run through more
   case statements on each iteration).
*/

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

const int TILE_WIDTH_GEMM = 16;
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

void multMat1( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ijk loop order. */
    for( i = 0; i < n; i++ )
        for( j = 0; j < n; j++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat2( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ikj loop order. */
    for( i = 0; i < n; i++ )
        for( k = 0; k < n; k++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat3( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jik loop order. */
    for( j = 0; j < n; j++ )
        for( i = 0; i < n; i++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat4( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jki loop order. */
    for( j = 0; j < n; j++ )
        for( k = 0; k < n; k++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat5( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kij loop order. */
    for( k = 0; k < n; k++ )
        for( i = 0; i < n; i++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat6( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kji loop order. */
    for( k = 0; k < n; k++ )
        for( j = 0; j < n; j++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

/* Question 1 */
// GPU based GEMM with SM-specific shared memory
__global__ void MatrixMultiplyKernel(const float* devM, const float* devN,float* devP, const int width){
	__shared__ float sM[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	__shared__ float sN[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = bx * TILE_WIDTH_GEMM + bx;
	int row = by * TILE_WIDTH_GEMM + ty;
	
	// Initialize accumulator to 0. Then multiply/add
	float pValue = 0;
	
	for (int m = 0; m < width / TILE_WIDTH_GEMM; m++) {
		sM[ty][tx] = devM[row *width+(m*TILE_WIDTH_GEMM + tx)];
		sN[ty][tx] = devN[col+(m *TILE_WIDTH_GEMM+ty)*width];
		__syncthreads();
		
		for (int k = 0; k < TILE_WIDTH_GEMM; ++k)
			pValue += sM[ty][k] * sN[k][tx];
		__syncthreads();
	}
	
	devP[row * width + col] = pValue;
}

/* Question 2 */
// Simple matrix copying
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// Matrix copy with shared memory
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// Native transpose
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// Coalesced transpose with block shared memory
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// Coalesced transpose with shared memory and matrix padding
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

/* uses timing features from sys/time.h that you haven't seen before */
int main( int argc, char **argv ) {
    // CPU implementation
    int nmax = 1000, i;

    void (*orderings[])(int,float *,float *,float *) =
        {&multMat1,&multMat2,&multMat3,&multMat4,&multMat5,&multMat6};
    char *names[] = {"ijk","ikj","jik","jki","kij","kji"};

    float *A = (float *)malloc( nmax*nmax * sizeof(float));
    float *B = (float *)malloc( nmax*nmax * sizeof(float));
    float *C = (float *)malloc( nmax*nmax * sizeof(float));

    struct timeval start, end;

    /* fill matrices with random numbers */
    for( i = 0; i < nmax*nmax; i++ ) A[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) B[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) C[i] = drand48()*2-1;

    for( i = 0; i < 6; i++) {
        /* multiply matrices and measure the time */
        gettimeofday( &start, NULL );
        (*orderings[i])( nmax, A, B, C );
        gettimeofday( &end, NULL );

        /* convert time to Gflop/s */
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        double Gflops = 2e-9*nmax*nmax*nmax/seconds;
        printf( "%s:\tn = %d, %.3f Gflop/s\n", names[i], nmax, Gflops );
    }
    
    free( A );
    free( B );
    free( C );
    
    printf("\n\n");
    
    // HW2: Question 1
    int m_size1 = 1600, n_size1 = 1600;
    int width1 = 1600;
    int iterations1 = 100;
    float ms1 = 0;
    float GFLOPs = 0;
    
    float *A_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));
    float *B_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));
    float *C_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));

    float *A_d1, *B_d1, *C_d1;
    hipMalloc((void**)&A_d1, m_size1*n_size1*sizeof(float));
    hipMalloc((void**)&B_d1, m_size1*n_size1*sizeof(float));
    hipMalloc((void**)&C_d1, m_size1*n_size1*sizeof(float));
    
    // events for timing
    hipEvent_t startEvent1, stopEvent1;
    checkCuda(hipEventCreate(&startEvent1));
    checkCuda(hipEventCreate(&stopEvent1));
	
    dim3 dimGrid1(width1/TILE_WIDTH_GEMM, width1/TILE_WIDTH_GEMM, 1);
    dim3 dimBlock1(TILE_WIDTH_GEMM, TILE_WIDTH_GEMM, 1);
    
    hipMemcpy(A_d1, A_h1, m_size1*n_size1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d1, B_h1, m_size1*n_size1*sizeof(float), hipMemcpyHostToDevice);
    
    // GPU based GEMM
    checkCuda( hipEventRecord(startEvent1, 0));
    for (int i = 0; i < iterations1; i++) {
        //gettimeofday( &start, NULL );
        MatrixMultiplyKernel<<<dimGrid1, dimBlock1>>>(A_d1, B_d1, C_d1, width1);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //GFLOPs += 2e-9*width1*width1*width1/seconds;
    }
    checkCuda( hipEventRecord(stopEvent1, 0) );
    checkCuda( hipEventSynchronize(stopEvent1) );
    checkCuda( hipEventElapsedTime(&ms1, startEvent1, stopEvent1) );
    printf( "GPU based GEMM time: %.3f ms\n", ms1 );
    hipMemcpy(C_h1, C_d1, m_size1*n_size1*sizeof(float), hipMemcpyDeviceToHost);
    GFLOPs = iterations1*2e-9*width1*width1*width1/(ms1*1e-3);
    
    printf( "GPU based GEMM: %.3f GFLOPs/s\n", GFLOPs );

    hipFree( A_d1 );
    hipFree( B_d1 );
    hipFree( C_d1 );
    free( A_h1 );
    free( B_h1 );
    free( C_h1 );

    printf("\n\n");
	
    // HW2: Question 2
    int m_size2 = 1024, n_size2 = 1024;
    int width2 = 1024;
    int iterations2 = 100;
    float Mem_Acc_Rate[5] = {0};
    float ms2 = 0;

    float *A_h2 = (float *)malloc( m_size2*n_size2*sizeof(float));
    float *B_h2 = (float *)malloc( m_size2*n_size2*sizeof(float));

    float *A_d2, *B_d2;
    hipMalloc((void**)&A_d2, m_size2*n_size2*sizeof(float));
    hipMalloc((void**)&B_d2, m_size2*n_size2*sizeof(float));
	
    // events for timing
    hipEvent_t startEvent2, stopEvent2;
    checkCuda(hipEventCreate(&startEvent2));
    checkCuda(hipEventCreate(&stopEvent2));
    
    dim3 dimGrid2(width2/TILE_DIM, width2/TILE_DIM, 1);
    dim3 dimBlock2(TILE_DIM, BLOCK_ROWS, 1);
	
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    
    // Simple matrix copying
    checkCuda( hipEventRecord(startEvent2, 0));
    for (int i = 0; i < iterations2; i++) {
        //gettimeofday( &start, NULL );
        copy<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //Mem_Acc_Rate[0] += 2*width2*width2*sizeof(float)/seconds/(float)(1e9);
    }
    checkCuda( hipEventRecord(stopEvent2, 0) );
    checkCuda( hipEventSynchronize(stopEvent2) );
    checkCuda( hipEventElapsedTime(&ms2, startEvent2, stopEvent2) );
    printf( "Simple matrix copying time: %.3f ms\n", ms2 );
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    Mem_Acc_Rate[0] = iterations2*2*width2*width2*sizeof(float)/(ms2*1e-3)/(float)(1e9);

    // Matrix copy with shared memory
    checkCuda( hipEventRecord(startEvent2, 0));
    for (int i = 0; i < iterations2; i++) {
        //gettimeofday( &start, NULL );
        copySharedMem<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //Mem_Acc_Rate[1] += 2*width2*width2*sizeof(float)/seconds/(float)(1e9);
    }
    checkCuda( hipEventRecord(stopEvent2, 0) );
    checkCuda( hipEventSynchronize(stopEvent2) );
    checkCuda( hipEventElapsedTime(&ms2, startEvent2, stopEvent2) );
    printf( "Matrix copy with shared memory time: %.3f ms\n", ms2 );
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    Mem_Acc_Rate[1] = iterations2*2*width2*width2*sizeof(float)/(ms2*1e-3)/(float)(1e9);
	
    // Native transpose
    checkCuda( hipEventRecord(startEvent2, 0));
    for (int i = 0; i < iterations2; i++) {
        //gettimeofday( &start, NULL );
        transposeNaive<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //Mem_Acc_Rate[2] += 2*width2*width2*sizeof(float)/seconds/(float)(1e9);
    }
    checkCuda( hipEventRecord(stopEvent2, 0) );
    checkCuda( hipEventSynchronize(stopEvent2) );
    checkCuda( hipEventElapsedTime(&ms2, startEvent2, stopEvent2) );
    printf( "Native transpose time: %.3f ms\n", ms2 );
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    Mem_Acc_Rate[2] = iterations2*2*width2*width2*sizeof(float)/(ms2*1e-3)/(float)(1e9);
	
    // Coalesced transpose with block shared memory
    checkCuda( hipEventRecord(startEvent2, 0));
    for (int i = 0; i < iterations2; i++) {
        //gettimeofday( &start, NULL );
        transposeCoalesced<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //Mem_Acc_Rate[3] += 2*width2*width2*sizeof(float)/seconds/(float)(1e9);
    }
    checkCuda( hipEventRecord(stopEvent2, 0) );
    checkCuda( hipEventSynchronize(stopEvent2) );
    checkCuda( hipEventElapsedTime(&ms2, startEvent2, stopEvent2) );
    printf( "Coalesced transpose with block shared memory time: %.3f ms\n", ms2 );
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    Mem_Acc_Rate[3] = iterations2*2*width2*width2*sizeof(float)/(ms2*1e-3)/(float)(1e9);
	
    // Coalesced transpose with shared memory and matrix padding
    checkCuda( hipEventRecord(startEvent2, 0));
    for (int i = 0; i < iterations2; i++) {
        //gettimeofday( &start, NULL );
        transposeNoBankConflicts<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        //gettimeofday( &end, NULL );
			
        //double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        //Mem_Acc_Rate[4] += 2*width2*width2*sizeof(float)/seconds/(float)(1e9);
    }
    checkCuda( hipEventRecord(stopEvent2, 0) );
    checkCuda( hipEventSynchronize(stopEvent2) );
    checkCuda( hipEventElapsedTime(&ms2, startEvent2, stopEvent2) );
    printf( "Coalesced transpose with shared memory and matrix padding time: %.3f ms\n", ms2 );
    hipMemcpy(A_d2, A_h2, m_size2*n_size2*sizeof(float), hipMemcpyHostToDevice);
    Mem_Acc_Rate[4] = iterations2*2*width2*width2*sizeof(float)/(ms2*1e-3)/(float)(1e9);

    printf("\n");
	
    printf( "Simple matrix copying: %.3f GB/s\n", Mem_Acc_Rate[0] );
    printf( "Matrix copy with shared memory: %.3f GB/s\n", Mem_Acc_Rate[1] );
    printf( "Native transpose: %.3f GB/s\n", Mem_Acc_Rate[2] );
    printf( "Coalesced transpose with block shared memory: %.3f GB/s\n", Mem_Acc_Rate[3] );
    printf( "Coalesced transpose with shared memory and matrix padding: %.3f GB/s\n", Mem_Acc_Rate[4] );

    hipFree( A_d2 );
    hipFree( B_d2 );
    free( A_h2 );
    free( B_h2 );
	
    return 0;
}
