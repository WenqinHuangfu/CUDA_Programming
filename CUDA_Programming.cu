#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* To save you time, we are including all 6 variants of the loop ordering
   as separate functions and then calling them using function pointers.
   The reason for having separate functions that are nearly identical is
   to avoid counting any extraneous processing towards the computation
   time.  This includes I/O accesses (printf) and conditionals (if/switch).
   I/O accesses are slow and conditional/branching statements could
   unfairly bias results (lower cases in switches must run through more
   case statements on each iteration).
*/

const int TILE_WIDTH_GEMM = 16;
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

void multMat1( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ijk loop order. */
    for( i = 0; i < n; i++ )
        for( j = 0; j < n; j++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat2( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is ikj loop order. */
    for( i = 0; i < n; i++ )
        for( k = 0; k < n; k++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat3( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jik loop order. */
    for( j = 0; j < n; j++ )
        for( i = 0; i < n; i++ )
            for( k = 0; k < n; k++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat4( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is jki loop order. */
    for( j = 0; j < n; j++ )
        for( k = 0; k < n; k++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat5( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kij loop order. */
    for( k = 0; k < n; k++ )
        for( i = 0; i < n; i++ )
            for( j = 0; j < n; j++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

void multMat6( int n, float *A, float *B, float *C ) {
    int i,j,k;
    /* This is kji loop order. */
    for( k = 0; k < n; k++ )
        for( j = 0; j < n; j++ )
            for( i = 0; i < n; i++ )
                C[i+j*n] += A[i+k*n]*B[k+j*n];
}

/* Question 1 */
// GPU based GEMM with SM-specific shared memory
__global__ void MatrixMultiplyKernel(const float* devM, const float* devN,float* devP, const int width){
	__shared__ float sM[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	__shared__ float sN[TILE_WIDTH_GEMM][TILE_WIDTH_GEMM];
	
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = bx * TILE_WIDTH_GEMM + bx;
	int row = by * TILE_WIDTH_GEMM + ty;
	
	// Initialize accumulator to 0. Then multiply/add
	float pValue = 0;
	
	for (int m = 0; m < width / TILE_WIDTH_GEMM; m++) {
		sM[ty][tx] = devM[row *width+(m*TILE_WIDTH_GEMM + tx)];
		sN[ty][tx] = devN[col+(m *TILE_WIDTH_GEMM+ty)*width];
		__syncthreads();
		
		for (int k = 0; k < TILE_WIDTH_GEMM; ++k)
			pValue += sM[ty][k] * sN[k][tx];
		__syncthreads();
	}
	
	devP[row * width + col] = pValue;
}

/* Question 2 */
// Simple matrix copying
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// Matrix copy with shared memory
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}

// Native transpose
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// Coalesced transpose with block shared memory
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// Coalesced transpose with shared memory and matrix padding
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

/* uses timing features from sys/time.h that you haven't seen before */
int main( int argc, char **argv ) {
    // CPU implementation
    int nmax = 1000,i;

    void (*orderings[])(int,float *,float *,float *) =
        {&multMat1,&multMat2,&multMat3,&multMat4,&multMat5,&multMat6};
    char *names[] = {"ijk","ikj","jik","jki","kij","kji"};

    float *A = (float *)malloc( nmax*nmax * sizeof(float));
    float *B = (float *)malloc( nmax*nmax * sizeof(float));
    float *C = (float *)malloc( nmax*nmax * sizeof(float));

    struct timeval start, end;

    /* fill matrices with random numbers */
    for( i = 0; i < nmax*nmax; i++ ) A[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) B[i] = drand48()*2-1;
    for( i = 0; i < nmax*nmax; i++ ) C[i] = drand48()*2-1;

    for( i = 0; i < 6; i++) {
        /* multiply matrices and measure the time */
        gettimeofday( &start, NULL );
        (*orderings[i])( nmax, A, B, C );
        gettimeofday( &end, NULL );

        /* convert time to Gflop/s */
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        double Gflops = 2e-9*nmax*nmax*nmax/seconds;
        printf( "%s:\tn = %d, %.3f Gflop/s\n", names[i], nmax, Gflops );
    }
    
    free( A );
    free( B );
    free( C );
    
    printf("\n\n");
    
    // HW2: Question 1
    int m_size1 = 1600, n_size1 = 1600;
    int width1 = 1600;
    int iterations1 = 100;
    float GFLOPs = 0;
    
    float *A_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));
    float *B_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));
    float *C_h1 = (float *)malloc( m_size1*n_size1*sizeof(float));

    float *A_d1, *B_d1, *C_d1;
    hipMalloc((void**)&A_d1, m_size1*n_size1*sizeof(float));
    hipMalloc((void**)&B_d1, m_size1*n_size1*sizeof(float));
    hipMalloc((void**)&C_d1, m_size1*n_size1*sizeof(float));
    
    dim3 dimGrid1(1, 1, 1);
    dim3 dimBlock1(1, 1, 1);
    
    hipMemcpy(A_d1, A_h1, m_size1*n_size1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d1, B_h1, m_size1*n_size1*sizeof(float), hipMemcpyHostToDevice);
		
    for (int i = 0; i < iterations1; i++) {
        gettimeofday( &start, NULL );
        MatrixMultiplyKernel<<<dimGrid1, dimBlock1>>>(A_d1, B_d1, C_d1, width1);
        gettimeofday( &end, NULL );
			
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        GFLOPs += 2e-9*width1*width1*width1/seconds;
    }
		
    hipMemcpy(C_h1, C_d1, m_size1*n_size1*sizeof(float), hipMemcpyDeviceToHost);

    GFLOPs /= iterations1;
		
    printf( "%.3f GFLOPs/s\n", GFLOPs );

    hipFree( A_d1 );
    hipFree( B_d1 );
    hipFree( C_d1 );
    free( A_h1 );
    free( B_h1 );
    free( C_h1 );

    printf("\n\n");
	
    // HW2: Question 2
    int m_size2 = 1024, n_size2 = 1024;
    int width2 = 1024;
    int iterations2 = 100;
    float Mem_Acc_Rate[5] = {0};

    float *A_h2 = (float *)malloc( m_size2*n_size2*sizeof(float));
    float *B_h2 = (float *)malloc( m_size2*n_size2*sizeof(float));

    float *A_d2, *B_d2;
    hipMalloc((void**)&A_d2, m_size2*n_size2*sizeof(float));
    hipMalloc((void**)&B_d2, m_size2*n_size2*sizeof(float));
    
    dim3 dimGrid2(1, 1, 1);
    dim3 dimBlock2(1, 1, 1);
    
    // Simple matrix copying
    for (int i = 0; i < iterations2; i++) {
        gettimeofday( &start, NULL );
        copy<<<dimGrid2, dimBlock2>>>(A_d2, B_d2);
        gettimeofday( &end, NULL );
			
        double seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
        Mem_Acc_Rate[0] += 2e-9*width1*width1*width1/seconds;
    }

    return 0;
}
